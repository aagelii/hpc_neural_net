#include "hip/hip_runtime.h"
#define USE_MNIST_LOADER
#define MNIST_DOUBLE
#include "mnist.h"
#include "layer.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <time.h>

static mnist_data *train_set, *test_set;
static unsigned int train_count, test_count;

// Define layers of CNN
static Layer layer_input = Layer(0, 0, 28 * 28);
static Layer layer_conv = Layer(5 * 5, 6, 24 * 24 * 6);
static Layer layer_sub = Layer(4 * 4, 1, 6 * 6 * 6);
static Layer layer_full = Layer(6 * 6 * 6, 10, 10);

// Loading the mnist dataset
static inline void loaddata()
{
	mnist_load("data/train-images.idx3-ubyte", "data/train-labels.idx1-ubyte",
		&train_set, &train_count);
	mnist_load("data/t10k-images.idx3-ubyte", "data/t10k-labels.idx1-ubyte",
		&test_set, &test_count);
}


// Forward propagation for a single image in the data set
static float forward_pass(double img[28][28])
{
	float input[28][28];

	// read img into input matrix
	for (int i = 0; i < 28; ++i) {
		for (int j = 0; j < 28; ++j) {
			input[i][j] = img[i][j];
		}
	}

	// reset to zero before doing forward pass
	layer_input.clear();
	layer_conv.clear();
	layer_sub.clear();
	layer_full.clear();

	// for timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	// load input image into first layer
	layer_input.load_image((float *)input);
	
	// forward pass for first layer
	fwd_pact_conv<<<64, 64>>>((float (*)[28])layer_input.output, (float (*)[24][24])layer_conv.pact, (float (*)[5][5])layer_conv.weight);
	fwd_add_bias_conv<<<64, 64>>>((float (*)[24][24])layer_conv.pact, layer_conv.bias);
	activation_function<<<64, 64>>>(layer_conv.pact, layer_conv.output, layer_conv.input_size);

	//forward pass for second layer
	fwd_pact_sub<<<64, 64>>>((float (*)[24][24])layer_conv.output, (float (*)[6][6])layer_sub.pact, (float (*)[4][4])layer_sub.weight);
	fwd_add_bias_sub<<<64, 64>>>((float (*)[6][6])layer_sub.pact, layer_sub.bias);
	activation_function<<<64, 64>>>(layer_sub.pact, layer_sub.output, layer_sub.input_size);

	// forward pass for output layer
	fwd_pact_full<<<64, 64>>>((float (*)[6][6])layer_sub.output, layer_full.pact, (float (*)[6][6][6])layer_full.weight);
	fwd_add_bias_full<<<64, 64>>>(layer_full.pact, layer_full.bias);
	activation_function<<<64, 64>>>(layer_full.pact, layer_full.output, layer_full.input_size);
	
	// for timing
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float ms;
	hipEventElapsedTime(&ms, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return ms;
}

// Backward propagation to update the weights
static float back_pass()
{
	// for timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	// back prop for output layer
	bkwd_weight_full<<<64, 64>>>((float (*)[6][6][6])layer_full.back_weight, layer_full.back_pact, (float (*)[6][6])layer_sub.output);
	bkwd_bias_full<<<64, 64>>>(layer_full.bias, layer_full.back_pact);

	// back prop for second layer
	bkwd_output_sub<<<64, 64>>>((float (*)[6][6])layer_sub.back_out, (float (*)[6][6][6])layer_full.weight, layer_full.back_pact);
	bkwd_pact_sub<<<64, 64>>>((float (*)[6][6])layer_sub.back_pact, (float (*)[6][6])layer_sub.back_out, (float (*)[6][6])layer_sub.pact);
	bkwd_weight_sub<<<64, 64>>>((float (*)[4][4])layer_sub.back_weight, (float (*)[6][6])layer_sub.back_pact, (float (*)[24][24])layer_conv.output);
	bkwd_bias_sub<<<64, 64>>>(layer_sub.bias, (float (*)[6][6])layer_sub.back_pact);

	// back prop for first layer
	bkwd_output_conv<<<64, 64>>>((float (*)[24][24])layer_conv.back_out, (float (*)[4][4])layer_sub.weight, (float (*)[6][6])layer_sub.back_pact);
	bkwd_pact_conv<<<64, 64>>>((float (*)[24][24])layer_conv.back_pact, (float (*)[24][24])layer_conv.back_out, (float (*)[24][24])layer_conv.pact);
	bkwd_weight_conv<<<64, 64>>>((float (*)[5][5])layer_conv.back_weight, (float (*)[24][24])layer_conv.back_pact, (float (*)[28])layer_input.output);
	bkwd_bias_conv<<<64, 64>>>(layer_conv.bias, (float (*)[24][24])layer_conv.back_pact);

	// apply the gradients to each layers weights/kernels
	grad<<<64, 64>>>(layer_full.weight, layer_full.back_weight, layer_full.size_kernel * layer_full.channels);
	grad<<<64, 64>>>(layer_sub.weight, layer_sub.back_weight, layer_sub.size_kernel * layer_sub.channels);
	grad<<<64, 64>>>(layer_conv.weight, layer_conv.back_weight, layer_conv.size_kernel * layer_conv.channels);

	// for timing
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float ms;
	hipEventElapsedTime(&ms, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return ms;
}

static void train()
{
	// for CUBLAS, creating a handle
	static hipblasHandle_t blas;
	hipblasCreate(&blas);

	float total_error;
	int iterations = 50;
	float total_time = 0.0;
	while (iterations-- > 0) {
		total_error = 0.0;
		for (int i = 0; i < train_count; ++i) {
			float cur_err = 0;
			total_time += forward_pass(train_set[i].data);

			// reset to zero before doing backward pass
			layer_full.backprop_clear();
			layer_sub.backprop_clear();
			layer_conv.backprop_clear();

			// Euclid distance of train_set[i]
			calc_error<<<10, 1>>>(layer_full.back_pact, layer_full.output, train_set[i].label, 10);
			// 2 norm
			hipblasSnrm2(blas, 10, layer_full.back_pact, 1, &cur_err);

			total_error += cur_err;

			total_time += back_pass();
		}
		total_error /= train_count;
		// output error and time on GPU
		fprintf(stdout, "Error: %e, GPU time: %lf\n", total_error, total_time);
		// stop training once threshold is passed
		if (total_error < threshold) {
			fprintf(stdout, "Finished Training!\n\n");
			break;
		}

	}
	
	fprintf(stdout, "\n Time - %lf\n", total_time);
}


// Prediction function that forward passes data getting its label
static unsigned int predict(double data[28][28])
{
	// to get the output layer
	float output[10];

	// do the forward pass
	forward_pass(data);

	// keep track of max prob in output array
	// start at 0 for label 0
	unsigned int cur_max = 0;

	// copy the forward pass output to host
	hipMemcpy(output, layer_full.output, sizeof(float) * 10, hipMemcpyDeviceToHost);

	// start at label 1
	for (int i = 1; i < 10; ++i) {
		if (output[cur_max] < output[i]) {
			cur_max = i;
		}
	}

	return cur_max;
}

// This function runs predictions on the test set to report test accuracy
static void run_test_set()
{
	// count hits
	int hit = 0;

	// iterate through the test set
	for (int i = 0; i < test_count; ++i) {
		if (predict(test_set[i].data) != test_set[i].label) {
			++hit;
		}
	}

	fprintf(stdout, "Accuracy: %.2lf%%\n", float(hit) / float(test_count) * 100.0);
}


int main()
{
	srand(time(NULL));
	std::cout<<"In main()!\n";
	hipError_t total_error = hipInit(0);
	if (total_error != hipSuccess) {
		fprintf(stderr, "CUDA initialisation failed with error code - %d\n", total_error);
		return 1;
	}

	std::cout<<"Loading data...\n";
	loaddata();
	std::cout<<"Beginning Training\n";
	train();
	std::cout<<"Running Test Set\n";
	run_test_set();

	return 0;
}